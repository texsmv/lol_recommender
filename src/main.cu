#include "scripts.h"
#include "structures.h"
#include "cud_defs.h"
#include "distances.h"
#include "knn.h"
#include "recomender.h"
#include <pthread.h>
#include "map.h"


int map_n_ratings, map_n_users, map_n_movies;
int n_ratings, n_users, n_movies;
int n_ratings_20, n_users_20, n_ratings_27, n_users_27, n_movies_27, n_ratings_l, n_users_l, n_movies_l, n_ratings_l2, n_users_l2, n_movies_l2, n_users_lol_17, n_ratings_lol_17;
int max_users = 8000;
int max_movies = 200;

map<int, map<int, float>* > map_users;
map<int, map<int, float>* > map_items;

float* values;
int *row_ind, * col_ind;
int * ind_users, *row_size;

float* d_values;
int *d_row_ind, * d_col_ind;
int * d_ind_users, * d_row_size;

thrust::device_vector< RBTree<int, float >* > d_map_users(max_users);




float* item_values;
int *item_row_ind, * item_col_ind;
int * ind_items, *item_row_size;

float* d_item_values;
int *d_item_row_ind, * d_item_col_ind;
int * d_ind_items, *d_item_row_size;


// para coseno ajustado
float *maxs, *mins, *averages;
float * d_averages;



vector<int> recomentar_equipo(  vector<int> ids_champions){
  RBTree<int, float>** d_n_map;
  create_map_cuda(ids_champions, d_n_map);

  reloj te;
  te.start();
  vector<int> recomendacion = knns_cosine(d_map_users, d_n_map, d_row_size, max_users, 5);
  te.stop();
  cout<<"Tiempo de prediccion: "<< te.time()<<endl;

  delete_map_cuda(d_n_map);
  return recomendacion;

}



int main(int argc, char const *argv[]) {

  vector<int> as = {1,2,3,4};
  vector<int> bs = {2,1,3,4};
  cout<<(as>bs)<<endl;


  n_ratings_27 = 27753444;
  n_users_27 = 283228;
  // n_movies_27 = 53889;

  n_ratings_20 = 20000263;
  n_users_20 = 138493;

  n_ratings_l = 49;
  n_users_l = 8;

  n_ratings_l2 = 21;
  n_users_l2 = 5;

  n_users_lol_17 = 3117;
  n_ratings_lol_17 = 31170;

  // n_ratings = n_ratings_20;
  // n_users = n_users_20;

  n_ratings = n_ratings_lol_17;
  n_users = n_users_lol_17;
  // n_movies = n_movies_27;

  // n_ratings = n_ratings_l2;
  // n_users = n_users_l2;




  // n_ratings
  n_of_users("dataset/HeroPart2.csv", n_ratings, n_users, false);
  cout<<n_ratings<<" "<<n_users<<endl;


  d_values = cuda_array<float>(n_ratings);
  d_row_ind = cuda_array<int>(n_ratings);
  d_col_ind = cuda_array<int>(n_ratings);
  d_ind_users = cuda_array<int>(max_users);
  d_row_size = cuda_array<int>(max_users);

  d_averages = cuda_array<float>(max_users);

  d_item_values = cuda_array<float>(n_ratings);
  d_item_row_ind = cuda_array<int>(n_ratings);
  d_item_col_ind = cuda_array<int>(n_ratings);
  d_ind_items = cuda_array<int>(max_movies);
  d_item_row_size = cuda_array<int>(max_movies);






  // read_ML_movies("../databases/ml-20m/movies.csv", movies_names, true);
  // read_ML_ratings("../databases/ml-20m/ratings.csv", n_ratings, n_users, true, values, row_ind, col_ind, ind_users, row_size, "27");

  // read_ML_movies("../../collaborative_filtering/databases/ml-latest/movies.csv", movies_names, true);
  // read_ML_ratings("../collaborative_filtering/databases/ml-latest/ratings.csv", n_ratings, n_users, true, values, row_ind, col_ind, ind_users, row_size, "27");

  reloj a;
  a.start();
  // read_ML_ratings("dataset/ratings.csv", n_ratings, n_users, true, values, row_ind, col_ind, ind_users, row_size, "l");
  // read_ML_ratings_items("dataset/ratings.csv", n_ratings, n_users, max_movies, true,  item_values,  item_row_ind,  item_col_ind,  ind_items, item_row_size, "l");
  read_ML_ratings("dataset/HeroPart2.csv", n_ratings, n_users, max_users, false, values, row_ind, col_ind, ind_users, row_size, "_2017");
  read_ML_ratings_items("dataset/HeroPart2.csv", n_ratings, n_users, max_movies, false,  item_values,  item_row_ind,  item_col_ind,  ind_items, item_row_size, "_2017");
  a.stop();
  cout<<"Tiempo de carga de bd: "<<a.time()<<"ms"<<endl;
  // read_ML_ratings("../collaborative_filtering/databases/libro/ratings.csv", n_ratings, n_users, true, values, row_ind, col_ind, ind_users, row_size, "l");
  // read_ML_ratings_items("../collaborative_filtering/databases/libro/ratings.csv", n_ratings, n_users, max_movies, true,  item_values,  item_row_ind,  item_col_ind,  ind_items, item_row_size, "l");




  // average_per_user(values, ind_users, row_size, maxs, mins, averages, max_users);



  cuda_H2D<float>(values, d_values, n_ratings);
  cuda_H2D<int>(row_ind, d_row_ind, n_ratings);
  cuda_H2D<int>(col_ind, d_col_ind, n_ratings);
  cuda_H2D<int>(ind_users, d_ind_users, max_users);
  cuda_H2D<int>(row_size, d_row_size, max_users);

  cuda_H2D<float>(item_values, d_item_values, n_ratings);
  cuda_H2D<int>(item_row_ind, d_item_row_ind, n_ratings);
  cuda_H2D<int>(item_col_ind, d_item_col_ind, n_ratings);
  cuda_H2D<int>(ind_items, d_ind_items, max_movies);
  cuda_H2D<int>(item_row_size, d_item_row_size, max_movies);



  size_t* t = new size_t();
  hipDeviceGetLimit( t, hipLimitMallocHeapSize);
  cout<<*t<<endl;
  hipDeviceSetLimit(hipLimitMallocHeapSize, (*t)* 100);
  CHECK(hipDeviceSynchronize());
  hipDeviceGetLimit( t, hipLimitMallocHeapSize);
  cout<<*t<<endl;

  create_maps_device(d_map_users, d_values, d_row_ind, d_col_ind, d_ind_users, d_row_size, max_users);









// Esto es todo lo que se necesita para recomendar

  vector<int> ids_champions = {2, 8};


  vector<int> recomendacion = recomentar_equipo(ids_champions);

  for (size_t i = 0; i < recomendacion.size(); i++) {
    cout<<recomendacion[i]<<endl;
  }

// -----------------------------------------------
























  return 0;
}
